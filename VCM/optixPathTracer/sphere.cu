#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, radius, , );

rtDeclareVariable(float3, world_normal, attribute world_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tValue, attribute tValue, );
rtBuffer<float3, 1>              Aabb_buffer;

RT_PROGRAM void intersect(int primIdx)
{
	float3 v = ray.origin - center;
	float dv = dot(ray.direction, v); 
	float temp = dv * dv - (dot(v, v) - radius * radius);
	if (temp > 0)
	{
		
		float t = -dv - sqrt(temp);
		if (t <ray.tmin)
			t = -dv + sqrt(temp);
		if(rtPotentialIntersection(t)) 
		{
			float3 p = ray.origin + t * ray.direction;
			geometric_normal = shading_normal = world_normal = normalize(ray.origin + t * ray.direction - center);
			tValue = t;
			rtReportIntersection( 0 );
        }
	}
}

RT_PROGRAM void bounds (int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;

	aabb->m_min = center - make_float3(radius);
	aabb->m_max = center + make_float3(radius);

	Aabb_buffer[0] = aabb->m_min;
	Aabb_buffer[1] = aabb->m_max;
}