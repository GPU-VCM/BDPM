#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, radius, , );
//rtDeclareVariable(int, lgt_instance, , ) = {0};

rtDeclareVariable(float3, world_normal, attribute world_normal, ); 
//rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tValue, attribute tValue, );
rtBuffer<float3, 1>              Aabb_buffer;

RT_PROGRAM void intersect(int primIdx)
{
	//printf("TESTING SPHERE\n");
	float3 v = ray.origin - center;
	float dv = dot(ray.direction, v); 
	float temp = dv * dv - (dot(v, v) - radius * radius);
	if (temp > 0)
	{
		
		float t = fminf(-dv + sqrt(temp), -dv - sqrt(temp));
		//float3 end = ray.origin + t * ray.direction;(length(end - center) - radius)<0.01f &&
		if(rtPotentialIntersection(t)) 
		{
			float3 p = ray.origin + t * ray.direction;
			//printf("Intersected:%f %f %f\n", p.x, p.y, p.z);
			world_normal = normalize(ray.origin + t * ray.direction - center);
			tValue = t;
			rtReportIntersection( 0 );
        }
	}
}

RT_PROGRAM void bounds (int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;

	aabb->m_min = center - make_float3(radius);
	aabb->m_max = center + make_float3(radius);

	Aabb_buffer[0] = aabb->m_min;
	Aabb_buffer[1] = aabb->m_max;
	//printf("AABB\n");
}