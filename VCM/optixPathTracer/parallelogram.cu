#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );
//rtDeclareVariable(int, lgt_instance, , ) = {0};

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float, tValue, attribute tValue, );
//rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtBuffer<float3, 1>              Aabb_buffer;

RT_PROGRAM void intersect(int primIdx)
{
  float3 n = make_float3( plane );
  float dt = dot(ray.direction, n );
  float t = (plane.w - dot(n, ray.origin))/dt;
  if( t > ray.tmin && t < ray.tmax ) {
	  //printf("%f %f\n", ray.tmin, ray.tmax);
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - anchor;
    float a1 = dot(v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if( rtPotentialIntersection( t ) ) {
          shading_normal = geometric_normal = n;
          texcoord = make_float3(a1,a2,0);
		  tValue = t;
		  
          //lgt_idx = lgt_instance;
          rtReportIntersection( 0 );
        }
      }
    }
  }
  //printf("intersected:%d\n", primIdx);
}

RT_PROGRAM void bounds (int primIdx, float result[6])
{
  // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
	const float3 tv1  = v1 / dot( v1, v1 );
	const float3 tv2  = v2 / dot( v2, v2 );
	const float3 p00  = anchor;
	const float3 p01  = anchor + tv1;
	const float3 p10  = anchor + tv2;
	const float3 p11  = anchor + tv1 + tv2;
	const float  area = length(cross(tv1, tv2));
  
	optix::Aabb* aabb = (optix::Aabb*)result;
  
	if(area > 0.0f && !isinf(area)) 
	{
		aabb->m_min = fminf( fminf( p00, p01 ), fminf( p10, p11 ) );
		aabb->m_max = fmaxf( fmaxf( p00, p01 ), fmaxf( p10, p11 ) );
	} 
	else 
	{
		aabb->invalidate();
	}

  	Aabb_buffer[0] = aabb->m_min;
	Aabb_buffer[1] = aabb->m_max;
  //printf("bouding:%d\n", primIdx);
  
}

	//float r = sqrt(1.0f - u1u2.x * u1u2.x);
	//float phi = 2 * M_PI * u1u2.y;
	//float theta = M_PI * u1u2.x;
	//float3 dir = make_float3(cos(phi), cos(theta), sin(phi));
	//printf("%f %f %f\n", dir.x, dir.y, dir.z);